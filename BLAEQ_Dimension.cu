#include "hip/hip_runtime.h"
#include "BLAEQ_Dimension.h"
#include "BLAEQ_CUDA_Kernels.cu"
#include "hipsparse.h"
#include "hipsparse.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/remove.h>



BLAEQ_Dimension::BLAEQ_Dimension(int dim, int K, int N, double* M, hipsparseHandle_t* cusparseHandle) {
	Dimension = dim;
	L = compute_layer(N, K);
	hipMalloc(&P_Matrices, L * sizeof(void*));	//Allocating space for P_matrix pointers
	hipMalloc(&Bandwidths, L * sizeof(double));	//Allocating space for bandwidths
	N = N;
	K = K;
	hipMalloc(&Coarsest_Mesh, L * sizeof(void*));
	BLAEQ_Generate_P_Matrices_Dimension(P_Matrices, Coarsest_Mesh, M, cusparseHandle);
}

void BLAEQ_Dimension::BLAEQ_Generate_P_Matrices_Dimension(hipsparseSpMatDescr_t** P_Matrices, hipsparseSpVecDescr_t* coarsestMesh, double* original_mesh, hipsparseHandle_t* cusparseHandle) {

	std::cout << "Generating Prolongation matrix for dimension " << Dimension << " ...";
	double* M_i_d = original_mesh;
	int N_i_d = N;
	for (int i = 0; i < L; i++) {
		double bandwidth = _bandwidth_generator(M_i_d, N_i_d, K);
		Bandwidths[(L - 1) - i] = bandwidth; //Store bandwidths in reverse order so that the coarsest layer corresponds to Bandwidths[0], second layer corresponds to Bandwidths[1] and so forth.
		double* M_ip1_d;
		int N_ip1_d;
		_generate_P_matrix(M_i_d, N_i_d, bandwidth, P_Matrices[(L - 1) - i], M_ip1_d, N_ip1_d, cusparseHandle); //Store P_Matrices in reverse order just like bandwidths.
		M_i_d = M_ip1_d;
		N_i_d = N_ip1_d;
	}
	int* coarsest_mesh_indices;
	hipMalloc(&coarsest_mesh_indices, N_i_d * sizeof(int));
	for (int i = 0; i < N_i_d; i++) {
		coarsest_mesh_indices[i] = i;
	}
	hipsparseCreateSpVec(coarsestMesh, N_i_d, N_i_d, coarsest_mesh_indices, M_i_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

}

void BLAEQ_Dimension::BLAEQ_Query_Dimension(double min, double max, hipsparseSpVecDescr_t* input_result, hipsparseSpVecDescr_t* output_result) {


	for (int i = 0; i < L; i++) {
		_logical_in_range_judgement(min, max, input_result, output_result);
		hipsparseSpMatDescr_t* P_matrix = P_Matrices[i];
		_BLAEQ_SpMSpV(P_matrix, input_result, output_result);
		output_result = input_result;
		//CUDA_in_range(double q_min, double q_max, double relaxation, double* data, double* indices, double* result_data, double* result_indices, int size)

	}
}


/*
*
	Below are tools necessary for BLAEQ. These functions should not be called outside of BLAEQ.
*
*/

int BLAEQ_Dimension::compute_layer(int N, int k) {
	return log2(N) / log2(k) + 1;
}

double BLAEQ_Dimension::_bandwidth_generator(double* vector, int size, int K) {
	int bin_count = size / K;
	double bandwidth = _compute_range(vector, size);
	double epsilon = bandwidth / 1000;
	return bandwidth + epsilon;
}


double BLAEQ_Dimension::_compute_range(double* vector, int size) {
	double max_val = DBL_MIN;
	double min_val = DBL_MAX;
	for (int i = 0; i < size; i++) {
		if (*vector < min_val) {
			min_val = *vector;
		}
		if (*vector > max_val) {
			max_val = *vector;
		}
		vector++;
	}
	//return max_val - min_val;
	return max_val; //Setting the lowerbound to 0 manually, seems more logical.
}

void BLAEQ_Dimension::_generate_P_matrix(double* M_i_d, int M_i_d_length, double bandwidth, hipsparseSpMatDescr_t* P_matrix_csc_balanced, double* M_ip1_d, int M_ip1_length, hipsparseHandle_t* cusparseHandle) {
	//Initializing P_matrix inn COO format
	int P_row_count = M_i_d_length;
	int P_col_count = M_i_d_length;
	int P_nnz_count = M_i_d_length;
	hipsparseSpMatDescr_t* P_matrix_coo;

	double* P_data;
	int* P_rows;
	int* P_cols;

	hipMalloc(&P_data, P_nnz_count * sizeof(double));
	hipMalloc(&P_rows, P_row_count * sizeof(int));
	hipMalloc(&P_cols, P_col_count * sizeof(int));

	int NUM_BLOCKS = (int)ceil(M_i_d_length / NUM_THREADS);
	CUDA_generate_P_matrix_kernel << <NUM_BLOCKS, NUM_THREADS >> > (M_i_d, M_i_d_length, bandwidth, P_data, P_rows, P_cols);

	hipsparseCreateCoo(P_matrix_coo, P_row_count, P_col_count, P_nnz_count, P_rows, P_cols, P_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

	int P_index_count_csc = M_i_d_length;
	int P_indptr_count_csc = floor(*thrust::max_element(M_i_d, M_i_d + M_i_d_length) / bandwidth) + 1;
	int P_data_count_csc = M_i_d_length;

	//Constructing M_ip1_d
	double* M_ip1_d_local;
	hipMalloc(&M_ip1_d_local, P_indptr_count_csc * sizeof(double));

	for (int i = 0; i < P_indptr_count_csc; i++) {
		M_ip1_d_local[i] = i * bandwidth + bandwidth / 2;
	}
	M_ip1_d = M_ip1_d_local;
	M_ip1_length = P_indptr_count_csc;

	//Initializing P_matrix in CSC format
	double* P_data_csc;
	int* P_index_csc;
	int* P_indptr_csc;

	hipMalloc(&P_data_csc, P_data_count_csc * sizeof(double));
	hipMalloc(&P_index_csc, P_index_count_csc * sizeof(int));
	hipMalloc(&P_indptr_csc, P_indptr_count_csc * sizeof(int));

	//Note: Function 'hipsparseXcoo2csr' can also be used for converting COO to csc, since it basically just merges indexes into indptrs, whose principle are identical for CSR & CSC.
	hipsparseXcoo2csr(*cusparseHandle, P_cols, P_nnz_count, P_col_count, P_indptr_csc, HIPSPARSE_INDEX_BASE_ZERO); //Converting COO to CSC


	int* P_indptr_balanced;
	double* M_ip1_d_local_balanced;
	_balance_P_matrix(MAX_COUNT_PER_COL, P_index_count_csc, P_indptr_count_csc, P_indptr_csc, M_ip1_d_local, P_indptr_balanced, M_ip1_d_local_balanced);

	hipFree(M_ip1_d_local);
	hipFree(P_indptr_csc);

	hipsparseCreateCsc(P_matrix_csc_balanced, P_row_count, P_col_count, P_nnz_count, P_indptr_balanced, P_index_csc, P_data_csc, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

}
void BLAEQ_Dimension::_balance_P_matrix(int MAX_BIN_SIZE, int M_index_count, int M_indptr_count, int* M_indptr, double* V_data, int* M_indptr_balanced, double* V_data_balanced) {
	//The code below is used for balancing the P matrix, such that the largest column size does not exceed MAX_COUNT_PER_COL.
	/*
		This part of the logic is implemented as follows:
		1.	Copy the first element into balanced_P directly;
		2.	For later elements:
		3.		Compare the temporary element with its previous element;
		4.		If the gap is smaller than MAX_COUNT_PER_COL, write temporary element into balanced_P directly;
		5.		If the gap is larger, write prev + MAX_COUNT_PER_COL into balanced_P;
		6.		Update prev = prev + MAX_COUNT_PER_COL, offset += 1;
		7.		Goto step 3;
		8.	Repeat 1-7 until balanced P indptr is built.
		NOTE: the indexes and data of P is identical to balanced_P, no change is required.
	*/
	int* balanced_indptr_buffer;
	double* balancd_V_data_buffer;
	int balanced_bin_count_upperbound = M_indptr_count + (N / MAX_COUNT_PER_COL);

	hipMalloc(&balanced_indptr_buffer, balanced_bin_count_upperbound * sizeof(int));
	hipMalloc(&balancd_V_data_buffer, balanced_bin_count_upperbound * sizeof(double));

	int balanced_P_col_count_csc = M_index_count;
	int balanced_array_offset = 0;
	for (int i = 0; i < M_indptr_count; i++) {
		int tmp_csc_index = M_indptr[i];
		int prev_csc_index = 0;
		if (i == 0) {
			balanced_indptr_buffer[i + balanced_array_offset] = tmp_csc_index;
			prev_csc_index = tmp_csc_index;

			balancd_V_data_buffer[i + balanced_array_offset] = V_data[i];
		}
		else if (tmp_csc_index - prev_csc_index <= MAX_COUNT_PER_COL) {
			balanced_indptr_buffer[i + balanced_array_offset] = tmp_csc_index;
			prev_csc_index = tmp_csc_index;

			balancd_V_data_buffer[i + balanced_array_offset] = V_data[i];
		}
		else if (tmp_csc_index - prev_csc_index > MAX_COUNT_PER_COL) {
			while (tmp_csc_index - prev_csc_index > MAX_COUNT_PER_COL) {
				balanced_indptr_buffer[i + balanced_array_offset] = prev_csc_index + MAX_COUNT_PER_COL;

				balancd_V_data_buffer[i + balanced_array_offset] = V_data[i];

				balanced_array_offset += 1;
				prev_csc_index += MAX_COUNT_PER_COL;
			}
			balanced_indptr_buffer[i + balanced_array_offset] = tmp_csc_index;
			prev_csc_index = tmp_csc_index;
		}
		else {
			std::cerr << "WTF??? The program should never reach here. Error when calling function _generate_P_matrix()." << std::endl;
		}
	}
	int balanced_M_indptr_count = M_indptr_count + balanced_array_offset;

	int* result_balanced_indptr;
	double* result_V_data;


	hipMalloc(&result_balanced_indptr, balanced_M_indptr_count * sizeof(int));
	hipMalloc(&result_V_data, (balanced_M_indptr_count - 1) * sizeof(double)); //The '-1' operator is required since |indptr| = |col| + 1, and |V| = |col|.

	for (int i = 0; i < balanced_M_indptr_count; i++) {
		result_balanced_indptr[i] = balanced_indptr_buffer[i];
	}
	for (int i = 0; i < balanced_M_indptr_count - 1; i++) {
		result_V_data[i] = balancd_V_data_buffer[i];
	}

	hipFree(&balanced_indptr_buffer);
	hipFree(&balancd_V_data_buffer);
}

void BLAEQ_Dimension::_logical_in_range_judgement(double min, double max, hipsparseSpVecDescr_t *input, hipsparseSpVecDescr_t* output) {
	void* index;
	void* data;
	int64_t* size;
	int64_t* nnz;
	hipsparseIndexType_t* index_type;
	hipsparseIndexBase_t* index_base;
	hipDataType* data_type;

	hipsparseSpVecGet(*input, size, nnz, &index, &data, index_type, index_base, data_type);
	int* tmp_result_indexes;
	double* tmp_result_data;
	hipMalloc(&tmp_result_indexes, sizeof(int) * *size);
	hipMalloc(&tmp_result_data, sizeof(double) * *size);

	CUDA_in_range_kernel <<<NUM_BLOCKS, NUM_THREADS >>> (min, max, Bandwidths[i] / 2, data, index, *size, tmp_result_data, tmp_result_indexes);

	int* index_end_ptr = thrust::remove(tmp_result_indexes, tmp_result_indexes + *size, 0);
	double* data_end_ptr = thrust::remove(tmp_result_data, tmp_result_data + *size, 0.0);

	int64_t nnz_size = index_end_ptr - tmp_result_indexes;

	hipsparseCreateSpVec(output, *size, nnz_size, tmp_result_indexes, tmp_result_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	hipFree(input);
}

void BLAEQ_Dimension::_BLAEQ_SpMSpV(hipsparseSpMatDescr_t *P_matrix, hipsparseSpVecDescr_t *input_vec, hipsparseSpVecDescr_t *result_vec) {

	int64_t* row_count;
	int64_t* col_count;
	int64_t* nnz_count;
	void** indptr;
	void** indexes;
	void** data;
	hipsparseIndexType_t* indptr_type;
	hipsparseIndexType_t* indexes_type;
	hipsparseIndexBase_t* idx_base;
	hipDataType* dataType;
	hipsparseCscGet(*P_matrix, row_count, col_count, nnz_count, indptr, indexes, data, indptr_type, indexes_type, idx_base, dataType);


	void** vec_indexes;
	void** vec_data;
	int64_t* vec_size;
	int64_t* vec_nnz;
	hipsparseIndexType_t* vec_index_type;
	hipsparseIndexBase_t* vec_index_base;
	hipDataType* vec_data_type;

	hipsparseSpVecGet(*input_vec, vec_size, vec_nnz, vec_indexes, vec_data, vec_index_type, vec_index_base, vec_data_type);

	int* res_indexes;
	double* res_data;

	int raw_result_vec_size = MAX_COUNT_PER_COL * *col_count;
		
	hipMalloc(&res_data, raw_result_vec_size * sizeof(double));
	hipMalloc(&res_indexes, raw_result_vec_size * sizeof(int));

	CUDA_BLAEQ_SpMSpV_kernel(row_count, col_count, nnz_count, MAX_COUNT_PER_COL, (double*)data, (int*) indexes, (int*) indptr, vec_nnz, (double*) vec_data, (int*) vec_indexes, res_data, res_indexes);


	int* cleaned_indexes = thrust::remove(res_indexes, res_indexes + raw_result_vec_size, 0);
	double* cleaned_data = thrust::remove(res_data, res_data + raw_result_vec_size, 0.0);

	if (cleaned_indexes - res_indexes != cleaned_data - res_data) {
		std::cerr << "WTF" << std::endl;
	}

	int cleaned_res_size = cleaned_indexes - res_indexes;
	hipsparseCreateSpVec(result_vec, *row_count, cleaned_res_size, cleaned_indexes, cleaned_data, *vec_index_type, *vec_index_base, *vec_data_type);

	hipFree(vec_indexes);
	hipFree(vec_data);
}