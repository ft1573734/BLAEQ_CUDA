#include "hip/hip_runtime.h"
#include "BLAEQ_Dimension.h"
#include "hipsparse.h"
#include "hipsparse.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/remove.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>


BLAEQ_Dimension::BLAEQ_Dimension(int dim, int K, int N, double* M, hipsparseHandle_t* cusparseHandle) {
	Dimension = dim;
	L = _compute_layer(N, K);
	hipMalloc(&P_Matrices, L * sizeof(void*));	//Allocating space for P_matrix pointers
	hipMalloc(&Bandwidths, L * sizeof(double));	//Allocating space for bandwidths
	N = N;
	K = K;
	hipMalloc(&Coarsest_Mesh, L * sizeof(void*));
	BLAEQ_Generate_P_Matrices_Dimension(P_Matrices, Coarsest_Mesh, M, cusparseHandle);
}

void BLAEQ_Dimension::BLAEQ_Generate_P_Matrices_Dimension(hipsparseSpMatDescr_t** P_Matrices, hipsparseSpVecDescr_t* coarsestMesh, double* original_mesh, hipsparseHandle_t* cusparseHandle) {

	std::cout << "Generating Prolongation matrix for dimension " << Dimension << " ...";
	double* M_i_d = original_mesh;
	int N_i_d = N;
	for (int i = 0; i < L; i++) {
		double bandwidth = _bandwidth_generator(M_i_d, N_i_d, K);
		Bandwidths[(L - 1) - i] = bandwidth; //Store bandwidths in reverse order so that the coarsest layer corresponds to Bandwidths[0], second layer corresponds to Bandwidths[1] and so forth.
		double* M_ip1_d = NULL;
		int* N_ip1_d = NULL;

		double* balanced_M_ip1_d = NULL;
		hipsparseSpMatDescr_t* tmp_P_matrix = NULL;
		hipsparseSpMatDescr_t* balanced_P_matrix = NULL;
		kernel.Generate_P_Matrix(M_i_d, N_i_d, bandwidth, tmp_P_matrix, M_ip1_d, N_ip1_d, cusparseHandle);

		kernel.Balance_P_Matrix(tmp_P_matrix, balanced_P_matrix, M_ip1_d, balanced_M_ip1_d, N_ip1_d);

		P_Matrices[(L - 1) - i] = balanced_P_matrix;

		M_i_d = balanced_M_ip1_d;
		N_i_d = *N_ip1_d;
	}
	int* coarsest_mesh_indices;
	hipMalloc(&coarsest_mesh_indices, N_i_d * sizeof(int));
	for (int i = 0; i < N_i_d; i++) {
		coarsest_mesh_indices[i] = i;
	}
	hipsparseCreateSpVec(coarsestMesh, N_i_d, N_i_d, coarsest_mesh_indices, M_i_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);


}

void BLAEQ_Dimension::BLAEQ_Query_Dimension(double min, double max, hipsparseSpVecDescr_t* result) {

	hipsparseSpVecDescr_t* logical_result;
	hipsparseSpVecDescr_t* this_layer;
	hipsparseSpVecDescr_t* next_layer;

	this_layer = Coarsest_Mesh;
	for (int i = 0; i < L; i++) {
		kernel.In_Range(min, max, Bandwidths[Dimension] / 2, this_layer, logical_result);
		hipsparseSpMatDescr_t* P_matrix = P_Matrices[i];
		kernel.SpMSpV(P_matrix, logical_result, next_layer);
		this_layer = next_layer;

	}
	result = this_layer;
}


/*
*
	Below are tools necessary for BLAEQ. These functions should not be called outside of BLAEQ.
*
*/

int BLAEQ_Dimension::_compute_layer(int N, int k) {
	return log2(N) / log2(k) + 1;
}

double BLAEQ_Dimension::_bandwidth_generator(double* vector, int size, int K) {
	int bin_count = size / K;
	double bandwidth = _compute_range(vector, size);
	double epsilon = bandwidth / 1000;
	return bandwidth + epsilon;
}


double BLAEQ_Dimension::_compute_range(double* vector, int size) {
	double max_val = DBL_MIN;
	double min_val = DBL_MAX;
	for (int i = 0; i < size; i++) {
		if (*vector < min_val) {
			min_val = *vector;
		}
		if (*vector > max_val) {
			max_val = *vector;
		}
		vector++;
	}
	//return max_val - min_val;
	return max_val; //Setting the lowerbound to 0 manually, seems more logical.
}
/*
void BLAEQ_Dimension::_generate_P_matrix(double* M_i_d, int M_i_d_length, double bandwidth, hipsparseSpMatDescr_t* P_matrix_csc_balanced, double* M_ip1_d, int M_ip1_length, hipsparseHandle_t* cusparseHandle) {
	//Initializing P_matrix inn COO format
	int P_row_count = M_i_d_length;
	int P_col_count = M_i_d_length;
	int P_nnz_count = M_i_d_length;
	hipsparseSpMatDescr_t* P_matrix_coo;

	double* P_data;
	int* P_rows;
	int* P_cols;

	hipMalloc(&P_data, P_nnz_count * sizeof(double));
	hipMalloc(&P_rows, P_row_count * sizeof(int));
	hipMalloc(&P_cols, P_col_count * sizeof(int));

	//int NUM_BLOCKS = (int)ceil(M_i_d_length / NUM_THREADS);
	CUDA_generate_P_matrix_kernel <<<NUM_BLOCKS, NUM_THREADS >>> (M_i_d, M_i_d_length, bandwidth, P_data, P_rows, P_cols);
	//CUDA_generate_P_matrix(M_i_d, M_i_d_length, bandwidth, P_data, P_rows, P_cols, NUM_BLOCKS, NUM_THREADS);

	hipsparseCreateCoo(P_matrix_coo, P_row_count, P_col_count, P_nnz_count, P_rows, P_cols, P_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

	int P_index_count_csc = M_i_d_length;
	int P_indptr_count_csc = floor(*thrust::max_element(M_i_d, M_i_d + M_i_d_length) / bandwidth) + 1;
	int P_data_count_csc = M_i_d_length;

	//Constructing M_ip1_d
	double* M_ip1_d_local;
	hipMalloc(&M_ip1_d_local, P_indptr_count_csc * sizeof(double));

	for (int i = 0; i < P_indptr_count_csc; i++) {
		M_ip1_d_local[i] = i * bandwidth + bandwidth / 2;
	}
	M_ip1_d = M_ip1_d_local;
	M_ip1_length = P_indptr_count_csc;

	//Initializing P_matrix in CSC format
	double* P_data_csc;
	int* P_index_csc;
	int* P_indptr_csc;

	hipMalloc(&P_data_csc, P_data_count_csc * sizeof(double));
	hipMalloc(&P_index_csc, P_index_count_csc * sizeof(int));
	hipMalloc(&P_indptr_csc, P_indptr_count_csc * sizeof(int));

	//Note: Function 'hipsparseXcoo2csr' can also be used for converting COO to csc, since it basically just merges indexes into indptrs, whose principle are identical for CSR & CSC.
	hipsparseXcoo2csr(*cusparseHandle, P_cols, P_nnz_count, P_col_count, P_indptr_csc, HIPSPARSE_INDEX_BASE_ZERO); //Converting COO to CSC


	int* P_indptr_balanced;
	double* M_ip1_d_local_balanced;
	_balance_P_matrix(MAX_COUNT_PER_COL, P_index_count_csc, P_indptr_count_csc, P_indptr_csc, M_ip1_d_local, P_indptr_balanced, M_ip1_d_local_balanced);

	hipFree(M_ip1_d_local);
	hipFree(P_indptr_csc);

	hipsparseCreateCsc(P_matrix_csc_balanced, P_row_count, P_col_count, P_nnz_count, P_indptr_balanced, P_index_csc, P_data_csc, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

}


void BLAEQ_Dimension::_logical_in_range_judgement(double min, double max, hipsparseSpVecDescr_t *input, hipsparseSpVecDescr_t* output) {
	void* index;
	void* data;
	int64_t* size;
	int64_t* nnz;
	hipsparseIndexType_t* index_type;
	hipsparseIndexBase_t* index_base;
	hipDataType* data_type;

	hipsparseSpVecGet(*input, size, nnz, & index, &data, index_type, index_base, data_type);



	int* tmp_result_indexes;
	double* tmp_result_data;
	hipMalloc(&tmp_result_indexes, sizeof(int) * *size);
	hipMalloc(&tmp_result_data, sizeof(double) * *size);
	// CUDA_in_range_kernel(double q_min, double q_max, double relaxation, double* data, double* indices, int size, double* result_data, int* result_indices)

	CUDA_in_range_kernel <<<NUM_BLOCKS, NUM_THREADS >>> (min, max, Bandwidths[Dimension] / 2, (double*) data, (int*) index, *size, tmp_result_data, tmp_result_indexes);
	//CUDA_in_range(min, max, Bandwidths[Dimension] / 2, (double*)data, (int*)index, *size, tmp_result_data, tmp_result_indexes, NUM_BLOCKS, NUM_THREADS);

	int* index_end_ptr = thrust::remove(tmp_result_indexes, tmp_result_indexes + *size, 0);
	double* data_end_ptr = thrust::remove(tmp_result_data, tmp_result_data + *size, 0.0);

	int64_t nnz_size = index_end_ptr - tmp_result_indexes;

	hipsparseCreateSpVec(output, *size, nnz_size, tmp_result_indexes, tmp_result_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	hipFree(input);
}


*/