#include "hip/hip_runtime.h"
#include "BLAEQ_CUDA_Kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipsparse.h"
#include "hipsparse.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/remove.h>
#include <stdio.h>
#include <math.h>



/*
	CUDA Kernel Functions
*/
__global__ void CUDA_in_range_kernel(double q_min, double q_max, double relaxation, double* data, int* indices, int size, double* result_data, int* result_indices) 
{
	// int i = threadIdx.x;
	// c[i] = a[i] + b[i];
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (t_idx < size) {
		if (q_min <= data[t_idx] <= q_max) {
			result_data[t_idx] = data[t_idx];
			result_indices[t_idx] = indices[t_idx];
		}
		else {
			result_data[t_idx] = 0.0;
			result_indices[t_idx] = 0;
		}
	}
}

__global__ void CUDA_generate_P_matrix_kernel(double* M_i_d, int M_i_d_length, double bandwidth, double* data, int* row, int* col) 
{
	//int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
	//for (int i = t_idx; i < M_i_d_length; i += blockDim.x * gridDim.x) {
	//	int bin_index = floor(M_i_d[t_idx] / bandwidth);
	//	row[t_idx] = t_idx;
	//	col[t_idx] = bin_index;
	//	data[t_idx] = bandwidth * bin_index + bandwidth / 2;
	//}
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (t_idx < M_i_d_length) {
		int bin_index = floor(M_i_d[t_idx] / bandwidth);
		row[t_idx] = t_idx;
		col[t_idx] = bin_index;
		data[t_idx] = bandwidth * bin_index + bandwidth / 2;
	}
	//return cudaStatus;
}


__global__ void CUDA_BLAEQ_SpMSpV_kernel(int64_t* P_row_count, int64_t* P_col_count, int64_t* P_nnz, int MAX_COL_SIZE, double* P_data, int* P_indexes, int* P_indptr, int64_t* V_nnz, double* V_data, int* V_indexes, double* Res_data, int* Res_indexes) 
{

	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (t_idx < *P_col_count) {
		int tmp_col = V_indexes[t_idx];
		int P_start_index = P_indptr[tmp_col];
		int P_end_index = P_indptr[tmp_col + 1];
		int tmp_col_size = P_end_index - P_start_index;
		int result_arr_start_index = t_idx * MAX_COL_SIZE;
		for (int i = 0; i < tmp_col_size; i++) {
			Res_data[result_arr_start_index + i] = P_data[P_start_index + i] * V_data[t_idx];
			Res_indexes[result_arr_start_index + i] = P_indexes[P_start_index + i];
		}
		for (int i = tmp_col_size; i < MAX_COL_SIZE; i++) {
			Res_data[result_arr_start_index + i] = 0.0;
			Res_indexes[result_arr_start_index + i] = 0;
		}
	}
}

BLAEQ_CUDA_Kernel::BLAEQ_CUDA_Kernel(int input_COL_SIZE_THRESHOLD) {
	COL_SIZE_THRESHOLD = input_COL_SIZE_THRESHOLD;
	NUM_BLOCKS = 16;
	NUM_THREADS = 512;
	//DEBUG = true;

}

BLAEQ_CUDA_Kernel::BLAEQ_CUDA_Kernel(){

}


void BLAEQ_CUDA_Kernel::In_Range(double min, double max, double relaxation, hipsparseSpVecDescr_t* input, hipsparseSpVecDescr_t* output) {
	void* index;
	void* data;
	int64_t* size;
	int64_t* nnz;
	hipsparseIndexType_t* index_type;
	hipsparseIndexBase_t* index_base;
	hipDataType* data_type;

	hipsparseSpVecGet(*input, size, nnz, &index, &data, index_type, index_base, data_type);


	int* tmp_result_indexes;
	double* tmp_result_data;
	hipMalloc(&tmp_result_indexes, sizeof(int) * *size);
	hipMalloc(&tmp_result_data, sizeof(double) * *size);
	// CUDA_in_range_kernel(double q_min, double q_max, double relaxation, double* data, double* indices, int size, double* result_data, int* result_indices)

	CUDA_in_range_kernel <<<NUM_BLOCKS, NUM_THREADS >>> (min, max, relaxation, (double*)data, (int*)index, *size, tmp_result_data, tmp_result_indexes);


	int* index_end_ptr = thrust::remove(tmp_result_indexes, tmp_result_indexes + *size, 0);
	double* data_end_ptr = thrust::remove(tmp_result_data, tmp_result_data + *size, 0.0);

	int64_t nnz_size = index_end_ptr - tmp_result_indexes;

	hipsparseCreateSpVec(output, *size, nnz_size, tmp_result_indexes, tmp_result_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	hipFree(input);
}


void BLAEQ_CUDA_Kernel::Generate_P_Matrix(double* M_i_d, int M_i_d_length, double bandwidth, hipsparseSpMatDescr_t* P_matrix_csc, double** M_ip1_d, int *M_ip1_size, hipsparseHandle_t* cusparseHandle) {

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Initializing P_matrix inn COO format
	double* M_i_d_DRAM;
	cudaStatus = hipMalloc(&M_i_d_DRAM, M_i_d_length * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	hipMemcpy(M_i_d_DRAM, M_i_d, M_i_d_length * sizeof(double), hipMemcpyHostToDevice);

	thrust::device_ptr<double> thrust_ptr_DRAM = thrust::device_pointer_cast(M_i_d_DRAM);
	int P_col_count = floor(*thrust::max_element(thrust_ptr_DRAM, thrust_ptr_DRAM + M_i_d_length) / bandwidth) + 1;
	int P_nnz_count = M_i_d_length;
	int P_row_count = M_i_d_length;

	*M_ip1_size = P_col_count;

	// Allocate GPU buffers for three vectors (two input, one output)

	double* P_data;
	int* P_rows;
	int* P_cols;

	cudaStatus = hipMalloc(&P_data, P_nnz_count * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&P_rows, P_nnz_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&P_cols, P_nnz_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	NUM_BLOCKS = (M_i_d_length + NUM_THREADS - 1) / NUM_THREADS; // Basically equal to ceil(M_i_d_length / NUM_THREADS)
	CUDA_generate_P_matrix_kernel <<<NUM_BLOCKS, NUM_THREADS>>> (M_i_d_DRAM, M_i_d_length, bandwidth, P_data, P_rows, P_cols);
	free(M_i_d);


	hipsparseSpMatDescr_t P_matrix_coo = nullptr;
	hipsparseCreateCoo(&P_matrix_coo, P_row_count, P_col_count, P_nnz_count, P_rows, P_cols, P_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

	if (DEBUG) {
		int* debug_coo_rows = (int*)malloc(P_nnz_count * sizeof(int));
		int* debug_coo_cols = (int*)malloc(P_nnz_count * sizeof(int));
		double* debug_coo_data = (double*)malloc(P_nnz_count * sizeof(double));

		hipMemcpy(debug_coo_rows, P_rows, P_nnz_count * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(debug_coo_cols, P_cols, P_nnz_count * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(debug_coo_data, P_data, P_nnz_count * sizeof(double), hipMemcpyDeviceToHost);

		free(debug_coo_rows);
		free(debug_coo_cols);
		free(debug_coo_data);
	}


	hipDeviceSynchronize();
	// Converting COO to CSC

	double* P_data_csc = P_data;
	int* P_index_csc = P_rows;
	int* P_indptr_csc;

	cudaStatus = hipMalloc(&P_indptr_csc, (P_col_count + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//Note: Function 'hipsparseXcoo2csr' can also be used for converting COO to csc, since it basically just merges indexes into indptrs, whose principle are identical for CSR & CSC.
	hipsparseXcoo2csr(*cusparseHandle, P_cols, P_nnz_count, P_col_count, P_indptr_csc, HIPSPARSE_INDEX_BASE_ZERO); //Converting COO to CSC
	hipsparseCreateCsc(P_matrix_csc, P_row_count, P_col_count, P_nnz_count, P_indptr_csc, P_index_csc, P_data_csc, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

	if (DEBUG) {
		int* indexes_host_debug = (int*)malloc(P_nnz_count * sizeof(int));
		int* indptr_host_debug = (int*)malloc((P_col_count + 1) * sizeof(int));
		double* data_host_debug = (double*)malloc(P_nnz_count * sizeof(double));

		hipMemcpy(indexes_host_debug, P_index_csc, P_nnz_count * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(indptr_host_debug, P_indptr_csc, (P_col_count + 1) * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(data_host_debug, P_data_csc, P_nnz_count * sizeof(double), hipMemcpyDeviceToHost);

		// Pause here and debug

		free(indexes_host_debug);
		free(indptr_host_debug);
		free(data_host_debug);
	}


	hipDeviceSynchronize();

	//Constructing M_ip1_d, we not only need to construct the P_matrix, we also need to construct the next-layer vector.
	//double* M_ip1_d_local = (double*)malloc(M_ip1_d_size * sizeof(double));

	//for (int i = 0; i < M_ip1_d_size; i++) {
	//	M_ip1_d_local[i] = i * bandwidth + bandwidth / 2;
	//}

	//cudaStatus = hipMalloc(&M_ip1_d, M_ip1_d_size * sizeof(double));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//hipMemcpy(M_ip1_d, M_ip1_d_local, M_ip1_d_size * sizeof(double), hipMemcpyHostToDevice);
	//M_ip1_length = &M_ip1_d_size;

	//free(M_ip1_d_local);

	double* M_ip1_d_host = (double*)malloc(*M_ip1_size * sizeof(double));

	for (int i = 0; i < *M_ip1_size; i++) {
		M_ip1_d_host[i] = i * bandwidth + bandwidth / 2;
	}
	*M_ip1_d = M_ip1_d_host;


Error:

	hipsparseDestroySpMat(P_matrix_coo);

	hipFree(P_cols);
	// You MUST NOT free P_data & P_rows here, since they are used in P_matrix_csc as well.
	// DON'T hipFree(P_data);
	// DON'T hipFree(P_rows);

	hipFree(M_i_d_DRAM);
}

//void BLAEQ_CUDA_Kernels::Balance_P_Matrix(int MAX_BIN_SIZE, int M_index_count, int M_indptr_count, int* M_indptr, double* V_data, int* M_indptr_balanced, double* V_data_balanced) {
void BLAEQ_CUDA_Kernel::Balance_P_Matrix(hipsparseSpMatDescr_t original_P_matrix, hipsparseSpMatDescr_t* balanced_P_matrix, double* original_V, int original_V_size, double** balanced_V, int* balanced_V_size) {
	//The code below is used for balancing the P matrix, such that the largest column size does not exceed MAX_BIN_SIZE.
	/*
		This part of the logic is implemented as follows:
		1.	Copy the first element into balanced_P directly;
		2.	For later elements:
		3.		Compare the temporary element with its previous element;
		4.		If the gap is smaller than MAX_BIN_SIZE, write temporary element into balanced_P directly;
		5.		If the gap is larger, write prev + MAX_BIN_SIZE into balanced_P;
		6.		Update prev = prev + MAX_BIN_SIZE, offset += 1;
		7.		Goto step 3;
		8.	Repeat 1-7 until balanced P indptr is built.
		NOTE: the indexes and data of P is identical to balanced_P, no change is required.
	*/

	//Getting components of original_P_matrix

	int64_t row_count;
	int64_t col_count;
	int64_t nnz;
	void* indptr_receiver;
	void* indexes_receiver;
	void* data_receiver;
	hipsparseIndexType_t indptr_type;
	hipsparseIndexType_t index_type;
	hipsparseIndexBase_t idxBase;
	hipDataType data_type;

	//hipsparseSpMatGetSize(original_P_matrix, &row_count, &col_count, &nnz);

	hipsparseCscGet(original_P_matrix, &row_count, &col_count, &nnz, &indptr_receiver, &indexes_receiver, &data_receiver, &indptr_type, &index_type, &idxBase, &data_type);

	if (DEBUG) {
		int* indexes_host_debug = (int*)malloc(nnz * sizeof(int));
		int* indptr_host_debug = (int*)malloc((col_count + 1) * sizeof(int));
		double* data_host_debug = (double*)malloc(nnz * sizeof(double));

		hipMemcpy(indexes_host_debug, indexes_receiver, nnz * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(indptr_host_debug, indptr_receiver, (col_count + 1) * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(data_host_debug, data_receiver, nnz * sizeof(double), hipMemcpyDeviceToHost);

		// Pause here and debug

		free(indexes_host_debug);
		free(indptr_host_debug);
		free(data_host_debug);
	}

	int* indptr = (int*)indptr_receiver;
	int* indices = (int*)indexes_receiver;
	double* data = (double*)data_receiver;

	//We only need to update the indptr of P matrix and new V, so we only need two buffers.
	int balanced_bin_count_upperbound = std::ceil(col_count + nnz / COL_SIZE_THRESHOLD);

	int* balanced_indptr_buffer_host = (int*)malloc((balanced_bin_count_upperbound + 1) * sizeof(int));
	double* balancd_V_data_buffer_host = (double*)malloc(balanced_bin_count_upperbound * sizeof(double));

	//Since indptr is on CUDA, we need to fetch it to host.
	int* indptr_host = (int*)malloc((col_count + 1) * sizeof(int));
	hipMemcpy(indptr_host, indptr, (col_count + 1) * sizeof(int), hipMemcpyDeviceToHost);

	//int balanced_P_col_count_csc = *col_count;
	int balanced_array_offset = 0;
	int prev_csc_index = 0;
	for (int i = 0; i < col_count + 1; i++) {
		int tmp_csc_index = indptr_host[i];
		if (i == 0) {
			balanced_indptr_buffer_host[i + balanced_array_offset] = tmp_csc_index;
			prev_csc_index = tmp_csc_index;

			balancd_V_data_buffer_host[i + balanced_array_offset] = original_V[i];
		}
		else if (tmp_csc_index - prev_csc_index <= COL_SIZE_THRESHOLD) {
			balanced_indptr_buffer_host[i + balanced_array_offset] = tmp_csc_index;
			prev_csc_index = tmp_csc_index;

			balancd_V_data_buffer_host[i + balanced_array_offset] = original_V[i];
		}
		else if (tmp_csc_index - prev_csc_index > COL_SIZE_THRESHOLD) {
			while (tmp_csc_index - prev_csc_index > COL_SIZE_THRESHOLD) {
				balanced_indptr_buffer_host[i + balanced_array_offset] = prev_csc_index + COL_SIZE_THRESHOLD;

				balancd_V_data_buffer_host[i + balanced_array_offset] = original_V[i];

				balanced_array_offset += 1;
				prev_csc_index += COL_SIZE_THRESHOLD;
			}
			balanced_indptr_buffer_host[i + balanced_array_offset] = tmp_csc_index;
			prev_csc_index = tmp_csc_index;
		}
		else {
			std::cerr << "WTF??? The program should never reach here. Error when calling function _generate_P_matrix()." << std::endl;
		}
	}

	*balanced_V_size = col_count + balanced_array_offset;
	
	//Generating & returning balanced vector, notice that we return vectors on host instead of on device.
	double* balanced_V_local = (double*)malloc(*balanced_V_size * sizeof(double));
	memcpy(balanced_V_local, balancd_V_data_buffer_host, *balanced_V_size * sizeof(double));
	*balanced_V = balanced_V_local;

	//Generating & returning balanced_P_matrix
	int* balanced_indptr;
	hipMalloc(&balanced_indptr, (*balanced_V_size + 1) * sizeof(int));
	hipMemcpy(balanced_indptr, balanced_indptr_buffer_host, (*balanced_V_size + 1) * sizeof(int), hipMemcpyHostToDevice);



	//Here, balanced_V_size equals column count.
	hipsparseCreateCsc(balanced_P_matrix, row_count, *balanced_V_size, nnz, balanced_indptr, indices, data, indptr_type, index_type, idxBase, data_type);


	free(balancd_V_data_buffer_host);
	free(balanced_indptr_buffer_host);
	free(indptr_host);
}


void BLAEQ_CUDA_Kernel::SpMSpV(hipsparseSpMatDescr_t* P_matrix, hipsparseSpVecDescr_t* input_vec, hipsparseSpVecDescr_t* result_vec) {

	int64_t* row_count;
	int64_t* col_count;
	int64_t* nnz_count;
	void** indptr;
	void** indexes;
	void** data;
	hipsparseIndexType_t* indptr_type;
	hipsparseIndexType_t* indexes_type;
	hipsparseIndexBase_t* idx_base;
	hipDataType* dataType;
	hipsparseCscGet(*P_matrix, row_count, col_count, nnz_count, indptr, indexes, data, indptr_type, indexes_type, idx_base, dataType);


	void** vec_indexes;
	void** vec_data;
	int64_t* vec_size;
	int64_t* vec_nnz;
	hipsparseIndexType_t* vec_index_type;
	hipsparseIndexBase_t* vec_index_base;
	hipDataType* vec_data_type;

	hipsparseSpVecGet(*input_vec, vec_size, vec_nnz, vec_indexes, vec_data, vec_index_type, vec_index_base, vec_data_type);

	int* res_indexes;
	double* res_data;

	int raw_result_vec_size = COL_SIZE_THRESHOLD * *col_count;

	hipMalloc(&res_data, raw_result_vec_size * sizeof(double));
	hipMalloc(&res_indexes, raw_result_vec_size * sizeof(int));

	CUDA_BLAEQ_SpMSpV_kernel <<<NUM_BLOCKS, NUM_THREADS >>> (row_count, col_count, nnz_count, COL_SIZE_THRESHOLD, (double*)data, (int*)indexes, (int*)indptr, vec_nnz, (double*)vec_data, (int*)vec_indexes, res_data, res_indexes);
	//CUDA_BLAEQ_SpMSpV(row_count, col_count, nnz_count, MAX_COUNT_PER_COL, (double*)data, (int*)indexes, (int*)indptr, vec_nnz, (double*)vec_data, (int*)vec_indexes, res_data, res_indexes, NUM_BLOCKS, NUM_THREADS);

	int* cleaned_indexes = thrust::remove(res_indexes, res_indexes + raw_result_vec_size, 0);
	double* cleaned_data = thrust::remove(res_data, res_data + raw_result_vec_size, 0.0);

	if (cleaned_indexes - res_indexes != cleaned_data - res_data) {
		std::cerr << "WTF" << std::endl;
	}

	int cleaned_res_size = cleaned_indexes - res_indexes;
	hipsparseCreateSpVec(result_vec, *row_count, cleaned_res_size, cleaned_indexes, cleaned_data, *vec_index_type, *vec_index_base, *vec_data_type);

	hipFree(vec_indexes);
	hipFree(vec_data);
}